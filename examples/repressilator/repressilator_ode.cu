#include "hip/hip_runtime.h"
#define NSPECIES 6
#define NPARAM 6
#define NREACT 12


__device__ double function_1(double a1,double a2,double a3,double a4){
    return a1 / (1 + pow(a2, a3)) + a4;
}

struct myFex{
    __device__ void operator()(int *neq, double *t, double *y, double *ydot/*, void *otherData*/)
    {
        int tid = blockDim.x * blockIdx.x + threadIdx.x;



        ydot[0]=(1.0*(tex2D(param_tex,0,tid)*function_1(tex2D(param_tex,4,tid),y[5],tex2D(param_tex,2,tid),tex2D(param_tex,1,tid)))-1.0*(tex2D(param_tex,0,tid)*tex2D(param_tex,5,tid)*y[0]))/tex2D(param_tex,0,tid);
        ydot[1]=(1.0*(tex2D(param_tex,0,tid)*tex2D(param_tex,3,tid)*y[0])-1.0*(tex2D(param_tex,0,tid)*tex2D(param_tex,3,tid)*y[1]))/tex2D(param_tex,0,tid);
        ydot[2]=(1.0*(tex2D(param_tex,0,tid)*function_1(tex2D(param_tex,4,tid),y[1],tex2D(param_tex,2,tid),tex2D(param_tex,1,tid)))-1.0*(tex2D(param_tex,0,tid)*tex2D(param_tex,5,tid)*y[2]))/tex2D(param_tex,0,tid);
        ydot[3]=(1.0*(tex2D(param_tex,0,tid)*tex2D(param_tex,3,tid)*y[2])-1.0*(tex2D(param_tex,0,tid)*tex2D(param_tex,3,tid)*y[3]))/tex2D(param_tex,0,tid);
        ydot[4]=(1.0*(tex2D(param_tex,0,tid)*function_1(tex2D(param_tex,4,tid),y[3],tex2D(param_tex,2,tid),tex2D(param_tex,1,tid)))-1.0*(tex2D(param_tex,0,tid)*tex2D(param_tex,5,tid)*y[4]))/tex2D(param_tex,0,tid);
        ydot[5]=(1.0*(tex2D(param_tex,0,tid)*tex2D(param_tex,3,tid)*y[4])-1.0*(tex2D(param_tex,0,tid)*tex2D(param_tex,3,tid)*y[5]))/tex2D(param_tex,0,tid);

    }
};


 struct myJex{
    __device__ void operator()(int *neq, double *t, double *y, int ml, int mu, double *pd, int nrowpd/*, void *otherData*/){
        return; 
    }
};